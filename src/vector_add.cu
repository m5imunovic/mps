#include "hip/hip_runtime.h"
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    size_t size = 50000 * sizeof(float);
    // Allocate and initialize the host vectors
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vectors A and B and output vector C
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);


    // Copy the host input vectors A and B in host memory to the device input vectors in device memory
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();
    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device global memory
    err = hipFree(d_A);
    err = hipFree(d_B);
    err = hipFree(d_C);
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    printf("Done\n");
    return 0;
}